
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime_api.h>

#define HAS_KERNEL 1
#define SIZE 512

#if HAS_KERNEL
__global__ void mykernel(float* t){
  
  int id = blockDim.x * blockIdx.x + threadIdx.x;

  t[id] = id*10.0f;

}

__global__ void mykernel2(float* t){
  
  int id = blockDim.x * blockIdx.x + threadIdx.x;

  t[id] = t[id]*2;

}
#endif

int main(){

  float* test;
  float* test_h;
  hipError_t res;

  test_h = (float*)malloc(sizeof(float)*SIZE);

  for(int i = 0 ; i < SIZE ; i ++){
    test_h[i] = i;
  }

  res = hipMalloc((void**)&test, sizeof(float) * SIZE);

  printf("hipMalloc => : %d\n", res);

  res = hipMemcpy(test, test_h, sizeof(float) * SIZE , hipMemcpyHostToDevice);

  printf("hipMemcpy => : %d(H to D)\n", res);

#if HAS_KERNEL
  dim3 threads(512, 1, 1);
  dim3 blocks(1, 1, 1);

  mykernel<<<blocks, threads>>>(test);
  mykernel2<<<blocks, threads>>>(test);
#endif

  res = hipMemcpy(test_h, test, sizeof(float) * SIZE , hipMemcpyDeviceToHost);

  printf("hipMemcpy => : %d(D to H)\n", res);

  for(int i = 0 ; i < SIZE ; i ++){
    if(test_h[i] != i*10.0f*2){
      printf("Result check : Failed...\n");
      exit(-1);
      break;
    }
  }

  printf("Result check : OK!!\n");

  res = hipFree(test);

  printf("hipFree   => : %d\n", res);

  return 0;
}
