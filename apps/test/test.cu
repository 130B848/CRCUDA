
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

int main(){

  hipStream_t s;
  hipError_t res;

  res = hipStreamCreate(&s);
  
  printf("res : %d\n", res);

  res = hipStreamDestroy(s);

  printf("res : %d\n", res);

  return 0;
}
